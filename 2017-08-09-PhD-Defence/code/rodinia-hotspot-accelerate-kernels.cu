#include "hip/hip_runtime.h"
3.67:cc: initialise kernel table
3.67:cc: deleting persistent cache
3.68:cc: (3.5,"\168\ESC\183\202vZ\190-,.V\235\199\EM\196\149")
#include <accelerate_cuda.h>
extern "C" __global__ void transform(const Int64 shIn0_0, const double* __restrict__ arrIn0_0, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh_1 = tmp_1 % shOut_1;
const Int64 sh_0 = tmp_0 % shOut_0;
const Int64 sh0 = ({ assert(mod(({ const Int64 v0 = (Int64) 512;  ; sh_1 * v0 + sh_0; }), shIn0_0) >= 0 && mod(({ const Int64 v0 = (Int64) 512;  ; sh_1 * v0 + sh_0; }), shIn0_0) < shIn0_0); mod(({ const Int64 v0 = (Int64) 512;  ; sh_1 * v0 + sh_0; }), shIn0_0); });
const double x0 = arrIn0_0[sh0];

arrOut_0[ix] = x0;
}
}


3.68:cc: waiting for nvcc...
3.69:cc: queue: 1.551 s, execute: 1.550 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31232.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31232.cu
3.69:cc: persist/save: /home/dpr964/.accelerate/accelerate-cuda-0.15.0.0/cache/3.5/zr149zr196zrEMzr199zr235Vziz2cUzmzr190ZZvzr202zr183zrESCzr168
3.70:cc: entry function 'transform' used 25 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
6.05:cc: entry function 'transform' used 25 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
6.09:cc: (3.5,"\210\136\ETB\FS\219\204\248\168Eh07{V\189\t")
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_0, const Int64 shOut_0, Int64* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh0 = ({ assert(ix >= 0 && ix < shOut_0); ix; });
const Int64 v0 = (Int64) 0;
const Int64 v1 = (Int64) -1 + ((Int64) 2 - sh0);
const Word8 v2 = v0 == v1;
Int64 lv50;

if (v2) {
lv50 = shIn0_0;
} else {
const Word8 v3 = v0 == (Int64) -1 + v1;
Int64 lv40;

if (v3) {
lv40 = shIn0_1;
} else {
lv40 = v0;
}
lv50 = lv40;
}
arrOut_0[ix] = lv50;
}
}


6.09:cc: (3.5,"\179\a\166\146\183\187\160\ETX\249\168\CAN\227\129-`\232")
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shIn0_0, const Int64* __restrict__ arrIn0_0, const Int64 shOut_0, Int64* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh0 = ({ assert(ix >= 0 && ix < shOut_0); ix; });
const Word8 v0 = sh0 < shIn0_0;
Int64 lv10;

if (v0) {
lv10 = arrIn0_0[sh0];
} else {
lv10 = (Int64) 0;
}
arrOut_0[ix] = lv10;
}
}


6.09:cc: (3.5,"\167\179\178q\175B:\NUL\154\&68!s\206\&0\129")
#include <accelerate_cuda.h>
extern "C" __global__ void transform(const Int64 shIn0_0, const Int64* __restrict__ arrIn0_0, const Int64 shOut_0, Int64* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh_0 = ({ assert(ix >= 0 && ix < shOut_0); ix; });
const Int64 sh0 = sh_0;
const Int64 x0 = arrIn0_0[sh0];

arrOut_0[ix] = x0;
}
}


6.09:cc: (3.5,"\229\235\242\212\DC3X\US\t\145u\248\162\156\237\226(")
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shIn0_0, const Int64* __restrict__ arrIn0_0, const Int64 shOut_0, Int64* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh0 = ({ assert(ix >= 0 && ix < shOut_0); ix; });
const Int64 v0 = (Int64) 1 - shIn0_0;
const Word8 v1 = sh0 < v0;
Int64 lv30;

if (v1) {
lv30 = (Int64) 0;
} else {
const Int64 v2 = sh0 - v0;

lv30 = arrIn0_0[v2];
}
arrOut_0[ix] = lv30;
}
}


6.09:cc: (3.5,"\170#\RS\171\165\&4\128\255V\138\244\128S\145\202\204")
#include <accelerate_cuda.h>
extern "C" __global__ void transform(const Int64 shIn0_0, const Int64* __restrict__ arrIn0_0, const Int64 shOut_0, Int64* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh_0 = ({ assert(ix >= 0 && ix < shOut_0); ix; });
const Int64 sh0 = sh_0 + ((Int64) -1 + shIn0_0);
const Int64 x0 = arrIn0_0[sh0];

arrOut_0[ix] = x0;
}
}


6.11:cc: (3.5,"H\179P\218\163g\252\251\249\206\bS\241\224\155 ")
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_0, const Int64 shIn1_0, const Int64* __restrict__ arrIn1_0, const Int64 shIn2_0, const Int64* __restrict__ arrIn2_0, const Int64 shIn3_0, const Word8* __restrict__ arrIn3_0, const Int64 shIn4_0, const Word8* __restrict__ arrIn4_0, const Int64 shIn5_0, const Int64* __restrict__ arrIn5_0, const Int64 shIn6_1, const Int64 shIn6_0, const double* __restrict__ arrIn6_0, const Int64 shOut_1, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = blockDim.x * gridDim.x;
int ix;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Int64 v0 = sh1 * shIn0_0 + sh0;
const double v1 = arrIn0_0[v0];

arrOut_0[ix] = v1 + 1.4583333333333335e-7 / (0.5 * (1750000.0 * (5.0e-4 * ({
const Int64 v2 = (Int64) 0;

;
1.6e-2 / (double) ({
const Word8 v3 = v2 == shIn1_0;
Int64 lv40;

if (v3) {
lv40 = v2;
} else {
lv40 = arrIn1_0[v2];
}
;
lv40;
}) * (1.6e-2 / (double) ({
const Word8 v5 = v2 == shIn2_0;
Int64 lv60;

if (v5) {
lv60 = v2;
} else {
lv60 = arrIn2_0[v2];
}
;
lv60;
}));
})))) * (({
const Int64 v7 = sh1 * shIn6_0 + sh0;

;
arrIn6_0[v7];
}) + ((({
const Word8 v9 = sh1 < ({ const Int64 v8 = max((Int64) 0, shIn0_1 - min((Int64) 1, shIn0_1));  ; v8; });
double lv170;

if (v9) {
const Int64 v10 = sh1 + min((Int64) 1, shIn0_1);
const Int64 v11 = v10 * shIn0_0 + sh0;

lv170 = arrIn0_0[v11];
} else {
lv170 = (double) (Int64) ({
const Int64 v16 = ({ assert(mod(({ const Word8 v12 = (Int64) 0 == shIn1_0; Int64 lv140;  if (v12) { lv140 = (Int64) 0; } else { const int v13 = (Int64) 0;  lv140 = arrIn1_0[v13]; } ; sh0; }), shIn3_0) >= 0 && mod(({ const Word8 v12 = (Int64) 0 == shIn1_0; Int64 lv140;  if (v12) { lv140 = (Int64) 0; } else { const int v13 = (Int64) 0;  lv140 = arrIn1_0[v13]; } ; sh0; }), shIn3_0) < shIn3_0); mod(({ const Word8 v12 = (Int64) 0 == shIn1_0; Int64 lv140;  if (v12) { lv140 = (Int64) 0; } else { const int v13 = (Int64) 0;  lv140 = arrIn1_0[v13]; } ; sh0; }), shIn3_0); });

;
arrIn3_0[v16];
});
}
;
lv170;
}) + ({
const Word8 v18 = sh1 < (Int64) 1;
double lv260;

if (v18) {
lv260 = (double) (Int64) ({
const Int64 v23 = ({ assert(mod(({ const Word8 v19 = (Int64) 0 == shIn1_0; Int64 lv210;  if (v19) { lv210 = (Int64) 0; } else { const int v20 = (Int64) 0;  lv210 = arrIn1_0[v20]; } ; sh0; }), shIn3_0) >= 0 && mod(({ const Word8 v19 = (Int64) 0 == shIn1_0; Int64 lv210;  if (v19) { lv210 = (Int64) 0; } else { const int v20 = (Int64) 0;  lv210 = arrIn1_0[v20]; } ; sh0; }), shIn3_0) < shIn3_0); mod(({ const Word8 v19 = (Int64) 0 == shIn1_0; Int64 lv210;  if (v19) { lv210 = (Int64) 0; } else { const int v20 = (Int64) 0;  lv210 = arrIn1_0[v20]; } ; sh0; }), shIn3_0); });

;
arrIn3_0[v23];
});
} else {
const Int64 v24 = (Int64) -1 + sh1;
const Int64 v25 = v24 * shIn0_0 + sh0;

lv260 = arrIn0_0[v25];
}
;
lv260;
}) - (double) ({
const Word8 v27 = sh1 < (Int64) 1;
Int64 lv530;

if (v27) {
lv530 = (Int64) ({
const Int64 v32 = ({ assert(mod(({ const Word8 v28 = (Int64) 0 == shIn1_0; Int64 lv300;  if (v28) { lv300 = (Int64) 0; } else { const int v29 = (Int64) 0;  lv300 = arrIn1_0[v29]; } ; sh0; }), shIn4_0) >= 0 && mod(({ const Word8 v28 = (Int64) 0 == shIn1_0; Int64 lv300;  if (v28) { lv300 = (Int64) 0; } else { const int v29 = (Int64) 0;  lv300 = arrIn1_0[v29]; } ; sh0; }), shIn4_0) < shIn4_0); mod(({ const Word8 v28 = (Int64) 0 == shIn1_0; Int64 lv300;  if (v28) { lv300 = (Int64) 0; } else { const int v29 = (Int64) 0;  lv300 = arrIn1_0[v29]; } ; sh0; }), shIn4_0); });

;
arrIn4_0[v32];
});
} else {
const Int64 v33 = (Int64) -1 + sh1;
const Word8 v39 = v33 < ({ const Int64 v34 = (Int64) 0; const Word8 v35 = v34 == shIn1_0; Int64 lv360;  if (v35) { lv360 = v34; } else { lv360 = arrIn1_0[v34]; } ; (Int64) -2 + ({ const Word8 v37 = v34 == shIn2_0; Int64 lv380;  if (v37) { lv380 = v34; } else { lv380 = arrIn2_0[v34]; } ; lv380; }); });
Int64 lv520;

if (v39) {
const Int64 v46 = ({ assert(mod(({ const Int64 v40 = (Int64) 0; const Word8 v41 = v40 == shIn1_0; Int64 lv420;  if (v41) { lv420 = v40; } else { lv420 = arrIn1_0[v40]; } ; sh0 * ((Int64) -2 + ({ const Word8 v43 = v40 == shIn2_0; Int64 lv440;  if (v43) { lv440 = v40; } else { lv440 = arrIn2_0[v40]; } ; lv440; })) + v33; }), shIn5_0) >= 0 && mod(({ const Int64 v40 = (Int64) 0; const Word8 v41 = v40 == shIn1_0; Int64 lv420;  if (v41) { lv420 = v40; } else { lv420 = arrIn1_0[v40]; } ; sh0 * ((Int64) -2 + ({ const Word8 v43 = v40 == shIn2_0; Int64 lv440;  if (v43) { lv440 = v40; } else { lv440 = arrIn2_0[v40]; } ; lv440; })) + v33; }), shIn5_0) < shIn5_0); mod(({ const Int64 v40 = (Int64) 0; const Word8 v41 = v40 == shIn1_0; Int64 lv420;  if (v41) { lv420 = v40; } else { lv420 = arrIn1_0[v40]; } ; sh0 * ((Int64) -2 + ({ const Word8 v43 = v40 == shIn2_0; Int64 lv440;  if (v43) { lv440 = v40; } else { lv440 = arrIn2_0[v40]; } ; lv440; })) + v33; }), shIn5_0); });

lv520 = arrIn5_0[v46];
} else {
lv520 = (Int64) ({
const Int64 v51 = ({ assert(mod(({ const Word8 v47 = (Int64) 0 == shIn1_0; Int64 lv490;  if (v47) { lv490 = (Int64) 0; } else { const int v48 = (Int64) 0;  lv490 = arrIn1_0[v48]; } ; sh0; }), shIn4_0) >= 0 && mod(({ const Word8 v47 = (Int64) 0 == shIn1_0; Int64 lv490;  if (v47) { lv490 = (Int64) 0; } else { const int v48 = (Int64) 0;  lv490 = arrIn1_0[v48]; } ; sh0; }), shIn4_0) < shIn4_0); mod(({ const Word8 v47 = (Int64) 0 == shIn1_0; Int64 lv490;  if (v47) { lv490 = (Int64) 0; } else { const int v48 = (Int64) 0;  lv490 = arrIn1_0[v48]; } ; sh0; }), shIn4_0); });

;
arrIn4_0[v51];
});
}
lv530 = lv520;
}
;
lv530;
}) * v1) / ({
const Int64 v54 = (Int64) 0;

;
1.6e-2 / (double) ({
const Word8 v55 = v54 == shIn2_0;
Int64 lv560;

if (v55) {
lv560 = v54;
} else {
lv560 = arrIn2_0[v54];
}
;
lv560;
}) / (2.0 * (100.0 * (5.0e-4 * (1.6e-2 / (double) ({
const Word8 v57 = v54 == shIn1_0;
Int64 lv580;

if (v57) {
lv580 = v54;
} else {
lv580 = arrIn1_0[v54];
}
;
lv580;
})))));
}) + ((({
const Int64 v59 = max((Int64) 0, shIn0_0 - min((Int64) 1, shIn0_0));
const Word8 v60 = sh0 < v59;
double lv680;

if (v60) {
const Int64 v61 = sh0 + min((Int64) 1, shIn0_0);
const Int64 v62 = sh1 * shIn0_0 + v61;

lv680 = arrIn0_0[v62];
} else {
lv680 = (double) (Int64) ({
const Int64 v67 = ({ assert(mod(({ const Word8 v63 = (Int64) 0 == shIn2_0; Int64 lv650;  if (v63) { lv650 = (Int64) 0; } else { const int v64 = (Int64) 0;  lv650 = arrIn2_0[v64]; } ; sh1 * (Int64) 1 + (sh0 - v59); }), shIn3_0) >= 0 && mod(({ const Word8 v63 = (Int64) 0 == shIn2_0; Int64 lv650;  if (v63) { lv650 = (Int64) 0; } else { const int v64 = (Int64) 0;  lv650 = arrIn2_0[v64]; } ; sh1 * (Int64) 1 + (sh0 - v59); }), shIn3_0) < shIn3_0); mod(({ const Word8 v63 = (Int64) 0 == shIn2_0; Int64 lv650;  if (v63) { lv650 = (Int64) 0; } else { const int v64 = (Int64) 0;  lv650 = arrIn2_0[v64]; } ; sh1 * (Int64) 1 + (sh0 - v59); }), shIn3_0); });

;
arrIn3_0[v67];
});
}
;
lv680;
}) + ({
const Word8 v69 = sh0 < (Int64) 1;
double lv770;

if (v69) {
lv770 = (double) (Int64) ({
const Int64 v74 = ({ assert(mod(({ const Word8 v70 = (Int64) 0 == shIn2_0; Int64 lv720;  if (v70) { lv720 = (Int64) 0; } else { const int v71 = (Int64) 0;  lv720 = arrIn2_0[v71]; } ; sh1 * (Int64) 1 + sh0; }), shIn3_0) >= 0 && mod(({ const Word8 v70 = (Int64) 0 == shIn2_0; Int64 lv720;  if (v70) { lv720 = (Int64) 0; } else { const int v71 = (Int64) 0;  lv720 = arrIn2_0[v71]; } ; sh1 * (Int64) 1 + sh0; }), shIn3_0) < shIn3_0); mod(({ const Word8 v70 = (Int64) 0 == shIn2_0; Int64 lv720;  if (v70) { lv720 = (Int64) 0; } else { const int v71 = (Int64) 0;  lv720 = arrIn2_0[v71]; } ; sh1 * (Int64) 1 + sh0; }), shIn3_0); });

;
arrIn3_0[v74];
});
} else {
const Int64 v75 = (Int64) -1 + sh0;
const Int64 v76 = sh1 * shIn0_0 + v75;

lv770 = arrIn0_0[v76];
}
;
lv770;
}) - (double) ({
const Word8 v78 = sh0 < (Int64) 1;
Int64 lv1040;

if (v78) {
lv1040 = (Int64) ({
const Int64 v83 = ({ assert(mod(({ const Word8 v79 = (Int64) 0 == shIn2_0; Int64 lv810;  if (v79) { lv810 = (Int64) 0; } else { const int v80 = (Int64) 0;  lv810 = arrIn2_0[v80]; } ; sh1; }), shIn4_0) >= 0 && mod(({ const Word8 v79 = (Int64) 0 == shIn2_0; Int64 lv810;  if (v79) { lv810 = (Int64) 0; } else { const int v80 = (Int64) 0;  lv810 = arrIn2_0[v80]; } ; sh1; }), shIn4_0) < shIn4_0); mod(({ const Word8 v79 = (Int64) 0 == shIn2_0; Int64 lv810;  if (v79) { lv810 = (Int64) 0; } else { const int v80 = (Int64) 0;  lv810 = arrIn2_0[v80]; } ; sh1; }), shIn4_0); });

;
arrIn4_0[v83];
});
} else {
const Int64 v84 = (Int64) -1 + sh0;
const Word8 v90 = v84 < ({ const Int64 v85 = (Int64) 0; const Word8 v86 = v85 == shIn2_0; Int64 lv870;  if (v86) { lv870 = v85; } else { lv870 = arrIn2_0[v85]; } ; (Int64) -2 + ({ const Word8 v88 = v85 == shIn1_0; Int64 lv890;  if (v88) { lv890 = v85; } else { lv890 = arrIn1_0[v85]; } ; lv890; }); });
Int64 lv1030;

if (v90) {
const Int64 v97 = ({ assert(mod(({ const Int64 v91 = (Int64) 0; const Word8 v92 = v91 == shIn2_0; Int64 lv930;  if (v92) { lv930 = v91; } else { lv930 = arrIn2_0[v91]; } ; sh1 * ((Int64) -2 + ({ const Word8 v94 = v91 == shIn1_0; Int64 lv950;  if (v94) { lv950 = v91; } else { lv950 = arrIn1_0[v91]; } ; lv950; })) + v84; }), shIn5_0) >= 0 && mod(({ const Int64 v91 = (Int64) 0; const Word8 v92 = v91 == shIn2_0; Int64 lv930;  if (v92) { lv930 = v91; } else { lv930 = arrIn2_0[v91]; } ; sh1 * ((Int64) -2 + ({ const Word8 v94 = v91 == shIn1_0; Int64 lv950;  if (v94) { lv950 = v91; } else { lv950 = arrIn1_0[v91]; } ; lv950; })) + v84; }), shIn5_0) < shIn5_0); mod(({ const Int64 v91 = (Int64) 0; const Word8 v92 = v91 == shIn2_0; Int64 lv930;  if (v92) { lv930 = v91; } else { lv930 = arrIn2_0[v91]; } ; sh1 * ((Int64) -2 + ({ const Word8 v94 = v91 == shIn1_0; Int64 lv950;  if (v94) { lv950 = v91; } else { lv950 = arrIn1_0[v91]; } ; lv950; })) + v84; }), shIn5_0); });

lv1030 = arrIn5_0[v97];
} else {
lv1030 = (Int64) ({
const Int64 v102 = ({ assert(mod(({ const Word8 v98 = (Int64) 0 == shIn2_0; Int64 lv1000;  if (v98) { lv1000 = (Int64) 0; } else { const int v99 = (Int64) 0;  lv1000 = arrIn2_0[v99]; } ; sh1; }), shIn4_0) >= 0 && mod(({ const Word8 v98 = (Int64) 0 == shIn2_0; Int64 lv1000;  if (v98) { lv1000 = (Int64) 0; } else { const int v99 = (Int64) 0;  lv1000 = arrIn2_0[v99]; } ; sh1; }), shIn4_0) < shIn4_0); mod(({ const Word8 v98 = (Int64) 0 == shIn2_0; Int64 lv1000;  if (v98) { lv1000 = (Int64) 0; } else { const int v99 = (Int64) 0;  lv1000 = arrIn2_0[v99]; } ; sh1; }), shIn4_0); });

;
arrIn4_0[v102];
});
}
lv1040 = lv1030;
}
;
lv1040;
}) * v1) / ({
const Int64 v105 = (Int64) 0;

;
1.6e-2 / (double) ({
const Word8 v106 = v105 == shIn1_0;
Int64 lv1070;

if (v106) {
lv1070 = v105;
} else {
lv1070 = arrIn1_0[v105];
}
;
lv1070;
}) / (2.0 * (100.0 * (5.0e-4 * (1.6e-2 / (double) ({
const Word8 v108 = v105 == shIn2_0;
Int64 lv1090;

if (v108) {
lv1090 = v105;
} else {
lv1090 = arrIn2_0[v105];
}
;
lv1090;
})))));
}) + (80.0 - v1) / (5.0e-4 / (100.0 * ({
const Int64 v110 = (Int64) 0;

;
1.6e-2 / (double) ({
const Word8 v111 = v110 == shIn2_0;
Int64 lv1120;

if (v111) {
lv1120 = v110;
} else {
lv1120 = arrIn2_0[v110];
}
;
lv1120;
}) * (1.6e-2 / (double) ({
const Word8 v113 = v110 == shIn1_0;
Int64 lv1140;

if (v113) {
lv1140 = v110;
} else {
lv1140 = arrIn1_0[v110];
}
;
lv1140;
}));
}))))));
}
}


6.17:cc: waiting for nvcc...
6.18:cc: queue: 1.576 s, execute: 1.576 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31234.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31234.cu
6.18:cc: queue: 1.621 s, execute: 1.621 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31232.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31232.cu
6.18:cc: persist/save: /home/dpr964/.accelerate/accelerate-cuda-0.15.0.0/cache/3.5/zrtzr189Vz7bU70hEzr168zr248zr204zr219zrFSzrETBzr136zr210
6.18:cc: queue: 1.699 s, execute: 1.699 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31236.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31236.cu
6.18:cc: queue: 1.737 s, execute: 1.737 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31233.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31233.cu
6.18:cc: queue: 1.701 s, execute: 1.701 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31235.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31235.cu
6.18:cc: entry function 'generate' used 9 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
6.18:cc: waiting for nvcc...
6.18:cc: persist/save: /home/dpr964/.accelerate/accelerate-cuda-0.15.0.0/cache/3.5/zr129zrza0zr206szn86zr154zrNULZCBzr175qzr178zr179zr167
6.18:cc: queue: 1.842 s, execute: 1.842 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31237.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31237.cu
6.18:cc: entry function 'transform' used 8 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
6.18:cc: entry function 'generate' used 9 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
6.18:cc: waiting for nvcc...
6.18:cc: persist/save: /home/dpr964/.accelerate/accelerate-cuda-0.15.0.0/cache/3.5/zr204zr202zr145Szr128zr244zr138Vzr255zr128zrza4zr165zr171zrRSzhzr170
6.18:cc: entry function 'transform' used 10 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
6.19:cc: waiting for nvcc...
6.19:cc: persist/save: /home/dpr964/.accelerate/accelerate-cuda-0.15.0.0/cache/3.5/z20Uzr155zr224zr241Szrbzr206zr249zr251zr252gzr163zr218Pzr179H
6.19:cc: entry function 'generate' used 52 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 56.2% : 1152 threads over 36 warps in 12 blocks
8.30:cc: entry function 'generate' used 9 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
8.30:cc: entry function 'transform' used 8 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
8.30:cc: entry function 'generate' used 9 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
8.30:cc: entry function 'transform' used 10 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 16 blocks
8.30:cc: entry function 'generate' used 52 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 56.2% : 1152 threads over 36 warps in 12 blocks
10.12:cc: (3.5,"\186\DC1\202\b\165\&3\SIo\193\242\219W\230 \187\155")
#include <accelerate_cuda.h>
extern "C" __global__ void fold(const Int64 shIn0_1, const Int64 shIn0_0, const double* __restrict__ arrIn0_0, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
extern volatile __shared__ double sdata0[];
double x0;
double y0;
double z0;
const Int64 sh1 = shIn0_1;
const Int64 sh0 = shIn0_0;
const int numIntervals = sh1;
const int intervalSize = sh0;
int ix;
int seg;

if (intervalSize == 0 || numIntervals == 0) {
const int gridSize = blockDim.x * gridDim.x;

for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < shOut_0; ix += gridSize) {
arrOut_0[ix] = -Infinity;
}
return;
}
for (seg = blockIdx.x; seg < numIntervals; seg += gridDim.x) {
const int start = seg * intervalSize;
const int end = start + intervalSize;
const int n = min(end - start, blockDim.x);

if (threadIdx.x >= n)
return;

ix = start - (start & warpSize - 1);
if (ix == start || intervalSize > blockDim.x) {
ix += threadIdx.x;
if (ix >= start) {
y0 = arrIn0_0[ix];
}
if (ix + blockDim.x < end) {
x0 = arrIn0_0[ix + blockDim.x];
if (ix >= start) {
z0 = fmax(y0, x0);
y0 = z0;
} else {
y0 = x0;
}
}
for (ix += 2 * blockDim.x; ix < end; ix += blockDim.x) {
x0 = arrIn0_0[ix];
z0 = fmax(y0, x0);
y0 = z0;
}
} else {
y0 = arrIn0_0[start + threadIdx.x];
}
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 512 < n) {
x0 = sdata0[threadIdx.x + 512];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 256 < n) {
x0 = sdata0[threadIdx.x + 256];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 128 < n) {
x0 = sdata0[threadIdx.x + 128];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 64 < n) {
x0 = sdata0[threadIdx.x + 64];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x < 32) {
if (threadIdx.x + 32 < n) {
x0 = sdata0[threadIdx.x + 32];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 16 < n) {
x0 = sdata0[threadIdx.x + 16];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 8 < n) {
x0 = sdata0[threadIdx.x + 8];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 4 < n) {
x0 = sdata0[threadIdx.x + 4];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 2 < n) {
x0 = sdata0[threadIdx.x + 2];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 1 < n) {
x0 = sdata0[threadIdx.x + 1];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
}
if (threadIdx.x == 0) {
x0 = -Infinity;
z0 = fmax(y0, x0);
y0 = z0;
arrOut_0[seg] = y0;
}
}
}


10.12:cc: (3.5,":\242\187\236\247\207&)\136w\250\&7\178~tq")
#include <accelerate_cuda.h>
extern "C" __global__ void foldAll(const Int64 shIn0_0, const double* __restrict__ arrIn0_0, const Int64 shOut_0, double* __restrict__ arrOut_0)
{
extern volatile __shared__ double sdata0[];
double x0;
double y0;
double z0;
const Int64 sh0 = shIn0_0;
const int shapeSize = sh0;
const int gridSize = blockDim.x * gridDim.x;
int ix = blockDim.x * blockIdx.x + threadIdx.x;

if (ix < shapeSize) {
y0 = arrIn0_0[ix];
for (ix += gridSize; ix < shapeSize; ix += gridSize) {
x0 = arrIn0_0[ix];
z0 = fmax(y0, x0);
y0 = z0;
}
}
sdata0[threadIdx.x] = y0;
ix = min(shapeSize - blockIdx.x * blockDim.x, blockDim.x);
__syncthreads();
if (threadIdx.x + 512 < ix) {
x0 = sdata0[threadIdx.x + 512];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 256 < ix) {
x0 = sdata0[threadIdx.x + 256];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 128 < ix) {
x0 = sdata0[threadIdx.x + 128];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 64 < ix) {
x0 = sdata0[threadIdx.x + 64];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x < 32) {
if (threadIdx.x + 32 < ix) {
x0 = sdata0[threadIdx.x + 32];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 16 < ix) {
x0 = sdata0[threadIdx.x + 16];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 8 < ix) {
x0 = sdata0[threadIdx.x + 8];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 4 < ix) {
x0 = sdata0[threadIdx.x + 4];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 2 < ix) {
x0 = sdata0[threadIdx.x + 2];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 1 < ix) {
x0 = sdata0[threadIdx.x + 1];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
}
if (threadIdx.x == 0) {
if (shapeSize > 0) {
if (gridDim.x == 1) {
x0 = -Infinity;
z0 = fmax(y0, x0);
y0 = z0;
}
arrOut_0[blockIdx.x] = y0;
} else {
arrOut_0[blockIdx.x] = -Infinity;
}
}
}


10.12:cc: (3.5,"\234\ETX\b\253. \197\213\130\130\182#Z\212Y^")
#include <accelerate_cuda.h>
extern "C" __global__ void foldAll(const Int64 shIn0_0, const double* __restrict__ arrIn0_0, const Int64 shOut_0, double* __restrict__ arrOut_0, const Int64 shRec_0, const double* __restrict__ arrRec_0)
{
extern volatile __shared__ double sdata0[];
double x0;
double y0;
double z0;
const Int64 sh0 = shRec_0;
const int shapeSize = sh0;
const int gridSize = blockDim.x * gridDim.x;
int ix = blockDim.x * blockIdx.x + threadIdx.x;

if (ix < shapeSize) {
y0 = arrRec_0[ix];
for (ix += gridSize; ix < shapeSize; ix += gridSize) {
x0 = arrRec_0[ix];
z0 = fmax(y0, x0);
y0 = z0;
}
}
sdata0[threadIdx.x] = y0;
ix = min(shapeSize - blockIdx.x * blockDim.x, blockDim.x);
__syncthreads();
if (threadIdx.x + 512 < ix) {
x0 = sdata0[threadIdx.x + 512];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 256 < ix) {
x0 = sdata0[threadIdx.x + 256];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 128 < ix) {
x0 = sdata0[threadIdx.x + 128];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 64 < ix) {
x0 = sdata0[threadIdx.x + 64];
z0 = fmax(y0, x0);
y0 = z0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x < 32) {
if (threadIdx.x + 32 < ix) {
x0 = sdata0[threadIdx.x + 32];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 16 < ix) {
x0 = sdata0[threadIdx.x + 16];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 8 < ix) {
x0 = sdata0[threadIdx.x + 8];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 4 < ix) {
x0 = sdata0[threadIdx.x + 4];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 2 < ix) {
x0 = sdata0[threadIdx.x + 2];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 1 < ix) {
x0 = sdata0[threadIdx.x + 1];
z0 = fmax(y0, x0);
y0 = z0;
sdata0[threadIdx.x] = y0;
}
}
if (threadIdx.x == 0) {
if (shapeSize > 0) {
if (gridDim.x == 1) {
x0 = -Infinity;
z0 = fmax(y0, x0);
y0 = z0;
}
arrOut_0[blockIdx.x] = y0;
} else {
arrOut_0[blockIdx.x] = -Infinity;
}
}
}


10.13:cc: waiting for nvcc...
10.13:cc: queue: 1.596 s, execute: 1.595 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31234.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31234.cu
10.13:cc: queue: 1.674 s, execute: 1.674 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31236.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31236.cu
10.13:cc: queue: 1.954 s, execute: 1.954 s
     ... /usr/local/cuda-6.0/bin/nvcc -I /home/dpr964/tail/accelerate0.15-hsenv/.hsenv/cabal/share/x86_64-linux-ghc-7.8.4/accelerate-cuda-0.15.0.0/cubits -arch=sm_35 -cubin -o /tmp/accelerate-cuda-31232/dragon31232.cubin --disable-warnings -DNDEBUG -O3 -m64 /tmp/accelerate-cuda-31232/dragon31232.cu
10.13:cc: persist/save: /home/dpr964/.accelerate/accelerate-cuda-0.15.0.0/cache/3.5/zr155zr187z20Uzr230Wzr219zr242zr193ozrSI3zr165zrbzr202zrDC1zr186
10.14:cc: entry function 'fold' used 16 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 2 blocks
10.14:cc: waiting for nvcc...
10.14:cc: persist/save: /home/dpr964/.accelerate/accelerate-cuda-0.15.0.0/cache/3.5/qtz7eUzr178zrza7zr250wzr136ZRzazr207zr247zr236zr187zr242ZC
10.14:cc: entry function 'foldAll' used 14 registers, 0 bytes smem, 0 bytes lmem, 0 bytes cmem
     ... multiprocessor occupancy 100.0% : 2048 threads over 64 warps in 2 blocks
